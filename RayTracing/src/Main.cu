#include "hip/hip_runtime.h"
#include <iostream>
#include "Vec3.h"

#define print(x) std::cout << x 
#define println(x) std::cout << x << std::endl


__global__
void add(int n, Vec3* x, Vec3* y)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i += stride)
	{
		y[i] += x[i];
	}

}


int test()
{
	int N = 1 << 30; // 1M elements

	println(N);

	Vec3* x;
	Vec3* y;


	hipMallocManaged(&x, N * sizeof(Vec3));
	hipMallocManaged(&y, N * sizeof(Vec3));



	// Run kernel on 1M elements on the CPU
	add(N, x, y);
	//add<<<1, 256 >>> (N, x, y);


	hipDeviceSynchronize();

	// Free memory
	hipFree(x);
	hipFree(y);


	return 0;
}
